#include "hip/hip_runtime.h"
#include "mylib.h"
#include "mylib.cuh"


__global__ void kernel_seuillageGPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j)
{
	float Csum;
	int i, j, k, iFirst, jFirst;

	iFirst = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	jFirst = blockIdx.y*BLOCK_SIZE;

	i = iFirst + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = jFirst + threadIdx.y;

	float nr = 0;

nr=d_image_in[2+j*3+i*3*size_j]/sqrtf(d_image_in[0+j*3+i*3*size_j]*d_image_in[0+j*3+i*3*size_j]+d_image_in[1+j*3+i*3*size_j]*d_image_in[1+j*3+i*3*size_j]+d_image_in[2+j*3+i*3*size_j]*d_image_in[2+j*3+i*3*size_j]);

	if(nr > 0.7)
		d_image_out[1+j*3+i*3*size_j] = d_image_in[2+j*3+i*3*size_j];
	else
		d_image_out[1+j*3+i*3*size_j] = d_image_in[1+j*3+i*3*size_j]; 

	d_image_out[0+j*3+i*3*size_j] = d_image_in[0+j*3+i*3*size_j];
	d_image_out[2+j*3+i*3*size_j] = d_image_in[2+j*3+i*3*size_j];


}


Mat seuillageGPU( Mat in)
{
	hipError_t error;
	Mat out;
	out.create(in.rows,in.cols,CV_8UC3);
	
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	
	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_seuillageGPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	return out;
}

